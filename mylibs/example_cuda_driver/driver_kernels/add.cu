
#include <hip/hip_runtime.h>

extern "C"
{
    __global__ void addIntKernel(int* c, const int* a, const int* b)
    {
        int i = threadIdx.x;
        c[i] = a[i] + b[i];
    }

    __global__ void addFloatKernel(float* c, const float* a, const float* b)
    {
        int i = threadIdx.x;
        c[i] = a[i] + b[i];
    }
}